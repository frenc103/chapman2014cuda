#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void mykernel(){
	printf("Hello world from device block %i !\n",blockIdx.x);
} /* end kernel */

int main(void) 
{
        /* launch this kernel 10 times*/
	mykernel<<< 2 , 5 >>>();
 	hipDeviceSynchronize();
	printf("Hello World from Host\n");
	return 0;
} /* end main */

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void print5(unsigned int *a)
{
	int i;

	hiprand_init(&a[blockIdx.x], 0, 0, 
    /* finish this code to calculate c element-wise from a and b where each block calculates one element */
	for (i = 0; i < 5; i++)
	{
		printf("Block %i Rand %i - %i", blockIdx.x, i, rand_r(&a[blockIdx.x]));
	}
}


/* experiment with different values of N.  */
/* how large can it be? */
#define N 3

int main()
{
	unsigned int *a;
	unsigned int *d_a;
	int size = N * sizeof( int );

	/* allocate space for device copies of a, b, c */
	
	hipMalloc( (void **) &d_a, size );

	/* allocate space for host copies of a, b, c and setup input values */

	a = (unsigned int *)malloc( size );

	/* intializing a, b, c on host */
	
	for( int i = 0; i < N; i++ )
	{
		a[i] = rand();
	}

	/* copy inputs to device */
	
	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );

	/* launch the kernel on the GPU */
	/* finish this kernel launch with N blocks and 1 thread per block */
	print5<<<N,1>>>(d_a);


	free(a);
	hipFree( d_a );
	
	return 0;
} /* end main */
